#include "hip/hip_runtime.h"
/* 矩阵加法 */

/**
 * @author RoyenHeart
 * @since 2021.09.22
 */

#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include "cudaTools.h"

using namespace std;

__global__ void matrixCalAdd(float *A, float *B, float *C) {
    int i = blockDim.x * blockIdx.x + threadIdx.x; // 定位元素，使用一维
    
    C[i] = A[i] + B[i];
}

int main() {

    // 声明矩阵大小
    int n = 1<<12;
    int size = n*n;
    size_t actualSize = size * sizeof(float);

    // 分配主机内存
    float *matrixCpuA = (float*)malloc(actualSize);
    float *matrixCpuB = (float*)malloc(actualSize);
    float *matrixCpuC = (float*)malloc(actualSize);
    
    // 判断是否正确分配主机内存
    if (matrixCpuA == NULL || matrixCpuB == NULL || matrixCpuC == NULL) {
        fprintf(stderr,"failed to allocate memory for host\n");
        exit(EXIT_FAILURE);
    }

    // 初始化数据
    for (int i = 0; i < size; i++) {
        matrixCpuA[i] = (float)rand() / (float)RAND_MAX;
        matrixCpuB[i] = (float)rand() / (float)RAND_MAX;
    }

    // 分配设备内存
    float *matrixGpuD = NULL;
    float *matrixGpuE = NULL;
    float *matrixGpuF = NULL;
    hipMalloc(&matrixGpuD, actualSize);
    hipMalloc(&matrixGpuE, actualSize);
    hipMalloc(&matrixGpuF, actualSize);

    // 主机数据转移至设备
    hipMemcpy(matrixGpuD, matrixCpuA, actualSize, hipMemcpyHostToDevice);
    hipMemcpy(matrixGpuE, matrixCpuB, actualSize, hipMemcpyHostToDevice);

    // 声明使用的线程/线程块和线程块数
    int threadsPerBlock = 256;
    int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    // 从CPU调用核函数
    double start,end;
    start = getExecuteTime();
    matrixCalAdd<<<blocks,threadsPerBlock>>>(matrixGpuD,matrixGpuE,matrixGpuF);
    end = getExecuteTime();
    printf("Total execute time on GPU is %lfs\n",end-start);
    
    // 核函数计算完毕后从设备内存转移数据至主机
    hipMemcpy(matrixCpuC, matrixGpuF, actualSize, hipMemcpyDeviceToHost);

    // 检查错误，精度要求为1e-5
    for (int i = 0; i < size; i++) {
        if (fabs(matrixCpuA[i] + matrixCpuB[i] - matrixCpuC[i]) > 1e-5) {
            fprintf(stderr,"Answer Wrong!\n");
            exit(EXIT_FAILURE);
        }
    }
    printf("PASSWED!\n");

    // 释放主机内存
    free(matrixCpuA);
    free(matrixCpuB);
    free(matrixCpuC);

    // 释放设备内存
    hipFree(matrixGpuD);
    hipFree(matrixGpuE);
    hipFree(matrixGpuF);

    return 0;
}