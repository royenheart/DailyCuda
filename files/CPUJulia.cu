/**
 * @file CPUJulia.cu
 * @author RoyenHeart
 * @brief Using CUDA C to draw Julia in CPU
 * @version 0.1
 * @date 2022-03-15 
 */

#include "cpu_bitmap.h"

#define DIM 256

// functions declare

void kernel(unsigned char *ptr);
int julia(int x, int y);

// struct declare

struct hipComplex {
    float r;
    float i;
    hipComplex(float a, float b) : r(a) , i(b) {}
    float magnitude2(void) {return r * r + i * i;}
    hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

int main(int argc, char *argv) {
    CPUBitmap bitmap(DIM, DIM);
    unsigned char *ptr = bitmap.get_ptr();
    
    kernel(ptr);

    bitmap.display_and_exit();
}

void kernel(unsigned char *ptr) {
    for (int y = 0; y < DIM; y++) {
        for (int x = 0; x < DIM; x++) {
            int offset = x + y * DIM;

            int juliaValue = julia(x, y);
            ptr[offset * 4 + 0] = 255 * juliaValue;
            ptr[offset * 4 + 1] = 0;
            ptr[offset * 4 + 2] = 0;
            ptr[offset * 4 + 3] = 255;
        }
    }
}

int julia(int x, int y) {
    const float scale = 1.5f;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    hipComplex c(-0.8f, 0.156f);
    hipComplex a(jx, jy);

    int i = 0;
    for (i = 0; i < 200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000) {
            return 0;
        }
    }

    return 1;   
}